#include "hip/hip_runtime.h"
#include "kernal.cuh"
#define STB_IMAGE_IMPLEMENTATION
#include <stb_image.h>
#define STB_IMAGE_WRITE_IMPLEMENTATION
#define STBI_MSC_SECURE_CRT
#include <stb_image_write.h>
#include <iostream>
__global__ void HW1(uchar3 * d_out, uchar3 * d_in)
{
	int threadId = blockIdx.x * blockDim.x + threadIdx.x;
	//printf("trheadIdx is: %d \n", threadId);
	float color = .299f * d_in[threadId].x + .587f * d_in[threadId].y + .114f *  d_in[threadId].z;
	d_out[threadId] = d_in[threadId];
}

void color2gray(imageInfo* ii, uchar3 * h_out)
{
	uchar3 *d_in;
	uchar3*d_out;

	int numPixels = ii->resolution;
	// allocate memory on GPU for picture
	checkCudaErrors(hipMalloc((void**)&d_in, numPixels * sizeof(uchar3)));
	checkCudaErrors(hipMalloc((void**)&d_out, numPixels * sizeof(uchar3)));
	//make sure no memory is left laying around
	checkCudaErrors(hipMemset(d_out, 0, numPixels * sizeof(uchar3)));
	// cpy CPU data to GPU data
	checkCudaErrors(hipMemcpy(d_in, ii->image, numPixels * sizeof(uchar3), hipMemcpyHostToDevice));

	// launch the kernel
	HW1<<<ii->width, ii->height >>> (d_out, d_in);
	hipError_t cudaStatus;
	// Check for any errors launching the kernel
	cudaStatus = hipGetLastError();
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "Kernel launch failed: %s\n", hipGetErrorString(cudaStatus));
	}

	// hipDeviceSynchronize waits for the kernel to finish, and returns
	// any errors encountered during the launch.
	cudaStatus = hipDeviceSynchronize();
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipDeviceSynchronize returned error code %d after launching addKernel!\n", cudaStatus);
	}
	cudaStatus =  hipMemcpy(h_out, d_out, numPixels * sizeof(uchar3), hipMemcpyDeviceToHost);
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipMemcpy failed!");
	}
	//checkCudaErrors(hipMemcpy(h_out, d_out, resolution, hipMemcpyDeviceToHost));
	checkCudaErrors(hipFree(d_in));
	checkCudaErrors(hipFree(d_out));
}

bool readImage(const char * filename, imageInfo* ii)
{
	int width, height, channels_in_file;
	ii->image = stbi_load(filename, &width, &height, &channels_in_file, 0);
	if (ii->image == NULL)
	{
		std::cerr << "Failed to load Image at: " << filename << std::endl;
		return false;
	}
	int reselution = width * height;
	ii->height = height;
	ii->width = width;
	ii->resolution = height * width;
	return true;
}

void writeImage(const char* filename, imageInfo* ii, const uchar3 *h_out)
{
	int res = stbi_write_jpg(filename, ii->width, ii->height, 3, h_out, 0);
	if (res == 0)
	{
		std::cout << "Failed to write image file" << std::endl;
		return;
	}
	std::cout << "Write Image Successfully to: " << filename << std::endl;
}

void exec(const char * inputFile, const char * outputFile)
{
	// ��ȡͼƬ
	imageInfo* ii = new imageInfo();
	bool res = readImage(inputFile, ii);
	if (!res) return;
	// ��ͼƬ�ҶȻ�
	uchar3 *h_out = (uchar3*)malloc(sizeof(uchar3) * ii->height * ii->width);
	if (h_out == NULL)
	{
		std::cout << "Failed to malloc h_out space" << std::endl;
		return;
	}
	color2gray(ii,h_out);
	// ����Ҷ�ͼƬ
	writeImage(outputFile, ii, h_out);
	// �ͷſռ�
	free(ii);
	free(h_out);
	h_out = NULL;
	stbi_image_free(ii->image);
}