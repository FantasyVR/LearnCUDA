#include "hip/hip_runtime.h"
#include "kernal.cuh"
#define STB_IMAGE_IMPLEMENTATION
#include <stb_image.h>
#define STB_IMAGE_WRITE_IMPLEMENTATION
#define STBI_MSC_SECURE_CRT
#include <stb_image_write.h>
#include <iostream>
uchar3 *d_rgbImage, *d_blurImage;
unsigned char *d_red, *d_green, *d_blue;
unsigned char *d_blurRed, *d_blurGreen, *d_blurBlue;
float *d_filter;
void your_gaussian_blur(imageInfo* ii, unsigned char* h_blurImage, const float *const h_filter, size_t filterWidth)
{
	// Allocate GPU memories
	allocateMemoryAndCopyToGPU(ii, h_filter, filterWidth);
	
	// Step 1: ��RGB��ͨ���ֿ�
	separateChannels<<<ii->height,ii->width>>>(d_red, d_green, d_blue, d_rgbImage, ii->height, ii->width);
	hipDeviceSynchronize(); checkCudaErrors(hipGetLastError());
	// Step 2: ��ÿ��ͨ���ֱ����Blur����
	gaussian_blur << <ii->height, ii->width >> > (d_red, d_blurRed,ii->height,ii->width,d_filter,filterWidth);
	hipDeviceSynchronize(); checkCudaErrors(hipGetLastError());
	gaussian_blur << <ii->height, ii->width >> > (d_green, d_blurGreen, ii->height, ii->width, d_filter, filterWidth);
	hipDeviceSynchronize(); checkCudaErrors(hipGetLastError());
	gaussian_blur << <ii->height, ii->width >> > (d_blue, d_blurBlue, ii->height, ii->width, d_filter, filterWidth);
	hipDeviceSynchronize(); checkCudaErrors(hipGetLastError());
	// Step 3: ����ͨ���ϲ�
	recombineChannels << <ii->height, ii->width >> > (d_blurRed, d_blurGreen, d_blurBlue ,d_blurImage,ii->height, ii->width);
	hipDeviceSynchronize(); checkCudaErrors(hipGetLastError());
	// Step 4: ��GPU��blur��Image ���� CPU�ڴ���
	checkCudaErrors(hipMemcpy(h_blurImage, d_blurImage, ii->resolution * sizeof(uchar3), hipMemcpyDeviceToHost));
	
	// �ͷ��Դ�ռ�
	cleanGPUMemory();
}
void allocateMemoryAndCopyToGPU(imageInfo *ii, const float* const h_filter, const size_t filterWidth)
{
	int numPixels = ii->resolution;
	// allocate memory on GPU for picture
	checkCudaErrors(hipMalloc((void**)&d_rgbImage, numPixels * sizeof(uchar3)));
	checkCudaErrors(hipMalloc((void**)&d_blurImage, numPixels * sizeof(uchar3)));
	// Copy Image from CPU to GPU
	checkCudaErrors(hipMemcpy(d_rgbImage, ii->image, numPixels * sizeof(uchar3), hipMemcpyHostToDevice));
	checkCudaErrors(hipMemset(d_blurImage, 0, numPixels * sizeof(uchar3)));
	// allocate channels for image
	checkCudaErrors(hipMalloc((void**)&d_red, numPixels * sizeof(unsigned char)));
	checkCudaErrors(hipMalloc((void**)&d_green, numPixels * sizeof(unsigned char)));
	checkCudaErrors(hipMalloc((void**)&d_blue, numPixels * sizeof(unsigned char)));
	checkCudaErrors(hipMemset(d_red, 0, numPixels * sizeof(unsigned char)));
	checkCudaErrors(hipMemset(d_green, 0, numPixels * sizeof(unsigned char)));
	checkCudaErrors(hipMemset(d_blue, 0, numPixels * sizeof(unsigned char)));
	// allocate channels for blured image
	checkCudaErrors(hipMalloc((void**)&d_blurRed, numPixels * sizeof(unsigned char)));
	checkCudaErrors(hipMalloc((void**)&d_blurGreen, numPixels * sizeof(unsigned char)));
	checkCudaErrors(hipMalloc((void**)&d_blurBlue, numPixels * sizeof(unsigned char)));
	checkCudaErrors(hipMemset(d_blurRed, 0, numPixels * sizeof(unsigned char)));
	checkCudaErrors(hipMemset(d_blurGreen, 0, numPixels * sizeof(unsigned char)));
	checkCudaErrors(hipMemset(d_blurBlue, 0, numPixels * sizeof(unsigned char)));

	// Allocate memory for filter
	checkCudaErrors(hipMalloc((void**)&d_filter, filterWidth * filterWidth * sizeof(float)));
	checkCudaErrors(hipMemcpy(d_filter, h_filter, filterWidth * filterWidth * sizeof(float), hipMemcpyHostToDevice));
}
void cleanGPUMemory()
{
	// Free GPU memory
	checkCudaErrors(hipFree(d_rgbImage));
	checkCudaErrors(hipFree(d_blurImage));
	checkCudaErrors(hipFree(d_red));
	checkCudaErrors(hipFree(d_green));
	checkCudaErrors(hipFree(d_blue));
	checkCudaErrors(hipFree(d_blurRed));
	checkCudaErrors(hipFree(d_blurGreen));
	checkCudaErrors(hipFree(d_blurBlue));
	checkCudaErrors(hipFree(d_filter));
}
__global__ void separateChannels(unsigned char * const redChannel, unsigned char * const greenChannel, 
	unsigned char * const blueChannel, const uchar3 * const inputImageRGB, int numRows, int numCols)
{
	int idx = blockIdx.x * blockDim.x + threadIdx.x;
	redChannel[idx] = inputImageRGB[idx].x;
	greenChannel[idx] = inputImageRGB[idx].y;
	blueChannel[idx] = inputImageRGB[idx].z;
}

__global__ void recombineChannels(const unsigned char * const redChannel, const unsigned char * const greenChannel, 
	const unsigned char * const blueChannel, uchar3 * const outputImageRGB, int numRows, int numCols)
{
	int idx = blockIdx.x * blockDim.x + threadIdx.x;
	unsigned char red = redChannel[idx];
	unsigned char green = greenChannel[idx];
	unsigned char blue = blueChannel[idx];
	outputImageRGB[idx] = make_uchar3(red, green, blue);
}

__global__ void gaussian_blur(const unsigned char * const inputChannel, unsigned char * const outputChannel, 
	int numRows, int numCols, const float * const filter, const int filterWidth)
{
	// compute current row and cloumn index
	int row = blockIdx.x;
	int col = threadIdx.x;
	int currentTidx = row * blockDim.x + col;
	// find its neighbors' index 
	int left  = (col - filterWidth / 2) ; left = left < 0 ? 0 : left;
	int right = (col + filterWidth / 2) ; right = right < numCols? right : numCols;
	int up    = (row - filterWidth / 2) ; up = up < 0 ? 0 : up;
	int below = (row + filterWidth / 2) ; below = below < numRows ? below : numRows;

	for (size_t i = left; i < left + filterWidth && i <= right; i++)
		for (size_t j = up; j < up + filterWidth && j <= below; j++)
		{
			int tIdx = j * blockDim.x + i;
			int x = i - col;
			int y = j - row;
			int filterIdx = (y - 1) * filterWidth + x;
			filterIdx = 0 - filterIdx;
			outputChannel[currentTidx] += filter[filterIdx] * inputChannel[tIdx];
		} 
}

bool readImage(const char * filename, imageInfo* ii)
{
	int width, height, channels_in_file;
	ii->image = stbi_load(filename, &width, &height, &channels_in_file, 0);
	if (ii->image == NULL)
	{
		std::cerr << "Failed to load Image at: " << filename << std::endl;
		return false;
	}
	ii->height = height;
	ii->width = width;
	ii->resolution = height * width;
	return true;
}

void writeImage(const char* filename, imageInfo* ii, const unsigned char *h_blurImage)
{
	int res = stbi_write_jpg(filename, ii->width, ii->height, 3, h_blurImage, 0);
	if (res == 0)
	{
		std::cout << "Failed to write image file" << std::endl;
		return;
	}
	std::cout << "Write Image Successfully to: " << filename << std::endl;
}

void exec(const char * inputFile, const char * outputFile)
{
	// ��ȡͼƬ
	imageInfo* ii = new imageInfo();
	bool res = readImage(inputFile, ii);
	if (!res) return;
	unsigned char *h_out = (unsigned char*)malloc(sizeof(uchar3) * ii->resolution);
	if (h_out == NULL)
	{
		std::cout << "Failed to malloc h_out space" << std::endl;
		return;
	}
	float h_filter[] = { 0.0,0.2,0.0,
						0.2,0.2,0.2,
						0.0,0.2,0.0 };
	// Blur ͼƬ
	your_gaussian_blur(ii,h_out,h_filter,3);
	// ���� Blur ͼƬ
	writeImage(outputFile, ii, h_out);
	// �ͷſռ�
	free(ii);
	free(h_out);
	h_out = NULL;
}